#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>

#define BLOCK_SIZE 1
#define divideup(x,y) (1 + (((x) - 1) / (y)))

// Nearest lower power of 2
__device__ __inline__ uint flp2(uint x)
{
  x = x | (x >> 1);
  x = x | (x >> 2);
  x = x | (x >> 4);
  x = x | (x >> 8);
  x = x | (x >> 16);
  return x - (x >> 1);
}

__device__ void add_stack(uint3float1* d_stacks,
                          uint* d_nstacks,
                          const uint3float1 val,
                          const int maxN)
{
	int k;
	uint num = (*d_nstacks);
	if (num < maxN) //add new value
	{
		k = num++;
		while (k > 0 && val.val > d_stacks[k-1].val)
		{
			d_stacks[k] = d_stacks[k - 1];
			--k;
		}

		d_stacks[k] = val;
		*d_nstacks = num;
	}
	else if (val.val >= d_stacks[0].val) return;
	else //delete highest value and add new
	{
		k = 1;
		while (k < maxN && val.val < d_stacks[k].val)
		{
			d_stacks[k - 1] = d_stacks[k];
			k++;
		}
		d_stacks[k - 1] = val;
	}
}

__device__ float dist(const uchar* __restrict img, const uint3 size, const uint3 ref, const uint3 cmp, const int k){
  float diff(0);
  for (int z = 0; z < k; ++z)
    for (int y = 0; y < k; ++y)
		    for (int x = 0; x < k; ++x){
        int rx = max(0, min(x + ref.x, size.x - 1));
        int ry = max(0, min(y + ref.y, size.y - 1));
        int rz = max(0, min(z + ref.z, size.z - 1));
        int cx = max(0, min(x + cmp.x, size.x - 1));
        int cy = max(0, min(y + cmp.y, size.y - 1));
        int cz = max(0, min(z + cmp.z, size.z - 1));
        //printf("rx: %d ry: %d rz: %d cx: %d cy: %d cz: %d\n", rx, ry, rz, cx, cy, cz);
		      float tmp = (img[(rx) + (ry)*size.x + (rz)*size.x*size.y] - img[(cx) + (cy)*size.x + (cz)*size.x*size.y]);
		      diff += tmp*tmp;
		    }
  return diff;
}

__global__ void debug_output(const uint3float1* __restrict d_stacks,
                             const uint* __restrict d_nstacks, 
                             const int size,
                             uchar* out)
{
  int count = 0;
  for (int i = 0; i < size; i++){

    //if (d_nstacks[i] == 0){
    //  count++;
    //  //printf("For pixel %d number of matched patches: %d\n", i, d_nstacks[i]);
    //}
    //printf("For pixel %d number of matched patches: %u first one %d %d %d %f\n", i, d_nstacks[i], d_stacks[i*4].x, d_stacks[i*4].y, d_stacks[i*4].z, d_stacks[i*4].val);
  }
  //printf("Count: %d from total: %d\n", count, size);
}
__global__ void k_block_matching(const uchar* __restrict img,
                                 const uint3 size,
                                 const uint3 tsize,
                                 const Parameters params,
								                         uint3float1* d_stacks,
								                         uint* d_nstacks,
                                 uchar* out)
{

  for (int Idz = blockDim.z * blockIdx.z + threadIdx.z; Idz < tsize.z; Idz += blockDim.z*gridDim.z)
    for (int Idy = blockDim.y * blockIdx.y + threadIdx.y; Idy < tsize.y; Idy += blockDim.y*gridDim.y)
      for (int Idx = blockDim.x * blockIdx.x + threadIdx.x; Idx < tsize.x; Idx += blockDim.x*gridDim.x)
  {

      int x = Idx * params.step_size;
      int y = Idy * params.step_size;
      int z = Idz * params.step_size;
      if (x >= size.x || y >= size.y || z >= size.z || x < 0 || y < 0 || z < 0)
        return;

      int wxb = fmaxf(0, x - params.window_size); // window x begin
      int wyb = fmaxf(0, y - params.window_size); // window y begin
      int wzb = fmaxf(0, z - params.window_size); // window z begin
      int wxe = fminf(size.x - 1, x + params.window_size); // window x end
      int wye = fminf(size.y - 1, y + params.window_size); // window y end
      int wze = fminf(size.z - 1, z + params.window_size); // window z end

      uint3 ref = make_uint3(x, y, z);

      for (int wz = wzb; wz <= wze; wz++)
        for (int wy = wyb; wy <= wye; wy++)
          for (int wx = wxb; wx <= wxe; wx++){
            float w = dist(img, size, ref, make_uint3(wx, wy, wz), params.patch_size);
            
            if (w < params.sim_th){
              add_stack(&d_stacks[(Idx + (Idy + Idz* tsize.y)*tsize.x)*params.maxN],
                &d_nstacks[Idx + (Idy + Idz* tsize.y)*tsize.x],
                uint3float1(wx, wy, wz, w),
                params.maxN);
            }
          }
    }
    
}


void run_block_matching(const uchar* __restrict d_noisy_volume,
                        const uint3 size,
                        const uint3 tsize,
                        const Parameters params,
					                   uint3float1 *d_stacks,
					                   uint *d_nstacks,
                        uchar* out)
{
	dim3 block(16, 16, 1);
 //dim3 grid(size.x / block.x / params.step_size, size.y / block.y / params.step_size, 1);
 dim3 grid(20, 20, 1);

 // Debug verification
 std::cout << "Total number of reference patches " << (tsize.x*tsize.y*tsize.z) << std::endl;

	std::cout << "Grid x: " << grid.x << " y: " << grid.y << " z: " << grid.z << std::endl;
	std::cout << "Block x: " << block.x << " y: " << block.y << " z: " << block.z << std::endl;
	std::cout << "Warps per block: " << block.x * block.y * block.z / 32 << std::endl;
	std::cout << "Treads per block: " << block.x * block.y * block.z << std::endl;
 std::cout << "Total threads: " << block.x*block.y*block.z*grid.x*grid.y*grid.z << std::endl;

 k_block_matching << <grid, block >> >(d_noisy_volume,
                                       size,
                                       tsize,
                                       params,
                                       d_stacks,
                                       d_nstacks,
                                       out);

 hipDeviceSynchronize();
 checkCudaErrors(hipGetLastError());
}

__global__ void k_nstack_to_pow(const uint* __restrict d_nstacks, uint* d_nstacks_pow, const int size){
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < size; i += blockDim.x*gridDim.x){
    d_nstacks_pow[i] = flp2(d_nstacks[i]);
    printf("Original: %d Stripped: %d\n", d_nstacks[i], flp2(d_nstacks[i]));
  }
}

__global__ void k_gather_cubes(const uchar* __restrict img,
                               const uint3 size,
                               const Parameters params,
                               const uint3float1* __restrict d_stacks,
                               const uint* __restrict d_nstacks,
                               const uint array_size,
                               uchar* d_gathered4dstack,
                               uint* d_nstacks_pow)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < array_size; i += blockDim.x*gridDim.x){

    int start = d_nstacks[i];
    int num = d_nstacks_pow[i];
    // compute fancy index
    // Each thread computes it's own index
    int idx = i*params.maxN+start-num;
    uint3float1 ref = d_stacks[idx];

    for (int z = 0; z < params.patch_size; ++z)
      for (int y = 0; y < params.patch_size; ++y)
        for (int x = 0; x < params.patch_size; ++x){

          int rx = max(0, min(x + ref.x, size.x - 1));
          int ry = max(0, min(y + ref.y, size.y - 1));
          int rz = max(0, min(z + ref.z, size.z - 1));

          int img_idx = (rx) + (ry)*size.x + (rz)*size.x*size.y;
          int stack_idx = i*params.patch_size*params.patch_size*params.patch_size + (x)+(y + z*params.patch_size)*params.patch_size;
          
          d_gathered4dstack[stack_idx] = img[img_idx];
        }

  }
}

void gather_cubes(const uchar* __restrict img,
                  const uint3 size,
                  const Parameters params,
                  const uint3float1* __restrict d_stacks,
                  const uint* __restrict d_nstacks,
                  uchar* d_gathered4dstack,
                  uint* d_nstacks_pow,
                  int &gather_stack_sum) // TODO: remove debug
{
  // Convert all the numbers to the lowest power of two
  uint array_size = (size.x*size.y*size.z) / params.step_size;
  k_nstack_to_pow << <20, 1024 >> >(d_nstacks, d_nstacks_pow, array_size);
  checkCudaErrors(hipGetLastError());
  thrust::device_ptr<uint> dt_nstacks_pow = thrust::device_pointer_cast(d_nstacks_pow);
  uint sum = thrust::reduce(dt_nstacks_pow, dt_nstacks_pow + array_size);
  std::cout << "Sum of pathces: "<< sum << std::endl;

  gather_stack_sum = sum; // TODO: remove debug

  // Allocate memory for gathered stacksuchar
  checkCudaErrors(hipMalloc((void**)&d_gathered4dstack, sizeof(uchar)*(sum*params.patch_size*params.patch_size*params.patch_size)));
  std::cout << "Allocated " << sizeof(uchar)*(sum*params.patch_size*params.patch_size*params.patch_size) << " bytes for gathered4dstack" << std::endl;

  k_gather_cubes << < 20, 256 >> > (img, size, params, d_stacks, d_nstacks, sum, d_gathered4dstack, d_nstacks_pow);
  checkCudaErrors(hipGetLastError());
}