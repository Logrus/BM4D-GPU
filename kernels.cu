#include "hip/hip_runtime.h"
#include "kernels.cuh"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/remove.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <math.h>

__global__ void k_debug_lookup_stacks(uint3float1 * d_stacks, int total_elements){
  int a = 345;
  for (int i = 0; i < 15; ++i){
    a += i;
    printf("%i: %d %d %d %f\n", i, d_stacks[i].x, d_stacks[i].y, d_stacks[i].z, d_stacks[i].val);
  }

}

void __global__ k_debug_lookup_4dgathered_stack(float* gathered_stack4d){
  for (int i = 0; i < 64 * 3; ++i){

    if (!(i % 4)) printf("\n");
    if (!(i % 16)) printf("------------\n");
    if (!(i % 64)) printf("------------\n");
    printf("%f ", gathered_stack4d[i]);
  }
}
__global__ void k_debug_lookup_int(int* gathered_stack4d){
  for (int i = 0; i < 64 * 3; ++i){
    if (!(i % 4)) printf("\n");
    if (!(i % 16)) printf("------------\n");
    if (!(i % 64)) printf("------------\n");
    printf("%d ", gathered_stack4d[i]);
  }
}
void debug_kernel_int(int* tmp){
  k_debug_lookup_int << <1, 1 >> >(tmp);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}
void debug_kernel(float* tmp){
  k_debug_lookup_4dgathered_stack << <1, 1 >> >(tmp);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}

// Nearest lower power of 2
__device__ __inline__ uint flp2(uint x)
{
  x = x | (x >> 1);
  x = x | (x >> 2);
  x = x | (x >> 4);
  x = x | (x >> 8);
  x = x | (x >> 16);
  return x - (x >> 1);
}

__device__ void add_stack(uint3float1* d_stacks,
                          uint* d_nstacks,
                          const uint3float1 val,
                          const int maxN)
{
	int k;
	uint num = (*d_nstacks);
	if (num < maxN) //add new value
	{
		k = num++;
		while (k > 0 && val.val > d_stacks[k-1].val)
		{
			d_stacks[k] = d_stacks[k - 1];
			--k;
		}

		d_stacks[k] = val;
		*d_nstacks = num;
	}
	else if (val.val >= d_stacks[0].val) return;
	else //delete highest value and add new
	{
		k = 1;
		while (k < maxN && val.val < d_stacks[k].val)
		{
			d_stacks[k - 1] = d_stacks[k];
			k++;
		}
		d_stacks[k - 1] = val;
	}
}

__device__ float dist(const uchar* __restrict img, const uint3 size, const uint3 ref, const uint3 cmp, const int k){
  float diff(0);
  for (int z = 0; z < k; ++z)
    for (int y = 0; y < k; ++y)
		    for (int x = 0; x < k; ++x){
        int rx = max(0, min(x + ref.x, size.x - 1));
        int ry = max(0, min(y + ref.y, size.y - 1));
        int rz = max(0, min(z + ref.z, size.z - 1));
        int cx = max(0, min(x + cmp.x, size.x - 1));
        int cy = max(0, min(y + cmp.y, size.y - 1));
        int cz = max(0, min(z + cmp.z, size.z - 1));
        //printf("rx: %d ry: %d rz: %d cx: %d cy: %d cz: %d\n", rx, ry, rz, cx, cy, cz);
		      float tmp = (img[(rx) + (ry)*size.x + (rz)*size.x*size.y] - img[(cx) + (cy)*size.x + (cz)*size.x*size.y]);
		      diff += tmp*tmp;
		    }
  return diff;
}

__global__ void k_block_matching(const uchar* __restrict img,
                                 const uint3 size,
                                 const uint3 tsize,
                                 const Parameters params,
								                         uint3float1* d_stacks,
								                         uint* d_nstacks)
{

  for (int Idz = blockDim.z * blockIdx.z + threadIdx.z; Idz < tsize.z; Idz += blockDim.z*gridDim.z)
    for (int Idy = blockDim.y * blockIdx.y + threadIdx.y; Idy < tsize.y; Idy += blockDim.y*gridDim.y)
      for (int Idx = blockDim.x * blockIdx.x + threadIdx.x; Idx < tsize.x; Idx += blockDim.x*gridDim.x)
  {

      int x = Idx * params.step_size;
      int y = Idy * params.step_size;
      int z = Idz * params.step_size;
      if (x >= size.x || y >= size.y || z >= size.z || x < 0 || y < 0 || z < 0)
        return;

      int wxb = fmaxf(0, x - params.window_size); // window x begin
      int wyb = fmaxf(0, y - params.window_size); // window y begin
      int wzb = fmaxf(0, z - params.window_size); // window z begin
      int wxe = fminf(size.x - 1, x + params.window_size); // window x end
      int wye = fminf(size.y - 1, y + params.window_size); // window y end
      int wze = fminf(size.z - 1, z + params.window_size); // window z end

      uint3 ref = make_uint3(x, y, z);

      for (int wz = wzb; wz <= wze; wz++)
        for (int wy = wyb; wy <= wye; wy++)
          for (int wx = wxb; wx <= wxe; wx++){
            float w = dist(img, size, ref, make_uint3(wx, wy, wz), params.patch_size);
            
            if (w < params.sim_th){
              add_stack(&d_stacks[(Idx + (Idy + Idz* tsize.y)*tsize.x)*params.maxN],
                &d_nstacks[Idx + (Idy + Idz* tsize.y)*tsize.x],
                uint3float1(wx, wy, wz, w),
                params.maxN);
            }
          }
    }
    
}

void run_block_matching(const uchar* __restrict d_noisy_volume,
                        const uint3 size,
                        const uint3 tsize,
                        const Parameters params,
					                   uint3float1 *d_stacks,
					                   uint *d_nstacks)
{
	dim3 block(32, 32, 1);
 //dim3 grid(size.x / block.x / params.step_size, size.y / block.y / params.step_size, 1);
 dim3 grid(20, 20, 1);

 // Debug verification
 std::cout << "Total number of reference patches " << (tsize.x*tsize.y*tsize.z) << std::endl;

 k_block_matching << <grid, block >> >(d_noisy_volume,
                                       size,
                                       tsize,
                                       params,
                                       d_stacks,
                                       d_nstacks);

 hipDeviceSynchronize();
 checkCudaErrors(hipGetLastError());
}

__global__ void k_nstack_to_pow(const uint* __restrict d_nstacks, uint* d_nstacks_pow, const int size){
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < size; i += blockDim.x*gridDim.x){
    if (i<size) 
      d_nstacks_pow[i] = flp2(d_nstacks[i]);
  }
}

__global__ void k_gather_cubes(const uchar* __restrict img,
                               const uint3 size,
                               const Parameters params,
                               const uint3float1* __restrict d_stacks,
                               const uint* __restrict d_nstacks,
                               const uint array_size,
                               float* d_gathered4dstack,
                               uint* d_nstacks_pow)
{
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < array_size; i += blockDim.x*gridDim.x){

    uint3float1 ref = d_stacks[i];
    int cube_size = params.patch_size*params.patch_size*params.patch_size;

    for (int z = 0; z < params.patch_size; ++z)
      for (int y = 0; y < params.patch_size; ++y)
        for (int x = 0; x < params.patch_size; ++x){

          int rx = max(0, min(x + ref.x, size.x - 1));
          int ry = max(0, min(y + ref.y, size.y - 1));
          int rz = max(0, min(z + ref.z, size.z - 1));

          int img_idx = (rx) + (ry)*size.x + (rz)*size.x*size.y;
          int stack_idx = i*cube_size + (x)+(y + z*params.patch_size)*params.patch_size;
          
          d_gathered4dstack[stack_idx] = img[img_idx];
        }

  }
}

struct is_not_empty
{
  __host__ __device__
    bool operator()(const uint3float1 x)
  {
    return (x.val != -1);
  }
};

void gather_cubes(const uchar* __restrict img,
                  const uint3 size,
                  const uint3 tsize,
                  const Parameters params,
                  uint3float1* d_stacks,
                  const uint* __restrict d_nstacks,
                  float* &d_gathered4dstack,
                  uint* d_nstacks_pow,
                  int &gather_stack_sum) 
{
  // Convert all the numbers to the lowest power of two
  uint array_size = (tsize.x*tsize.y*tsize.z);
  k_nstack_to_pow << <20, 1024 >> >(d_nstacks, d_nstacks_pow, array_size);
  checkCudaErrors(hipGetLastError());
  thrust::device_ptr<uint> dt_nstacks_pow = thrust::device_pointer_cast(d_nstacks_pow);
  uint sum = thrust::reduce(dt_nstacks_pow, dt_nstacks_pow + array_size);
  std::cout << "Sum of pathces: "<< sum << std::endl;

  gather_stack_sum = sum; 
   
  k_debug_lookup_stacks << <1, 1 >> >(d_stacks, tsize.x*tsize.y*tsize.z);

  // Make a compaction
  uint3float1 * d_stacks_compacted;
  checkCudaErrors(hipMalloc((void**)&d_stacks_compacted, sizeof(uint3float1)*(params.maxN *tsize.x*tsize.y*tsize.z)));
  thrust::device_ptr<uint3float1> dt_stacks = thrust::device_pointer_cast(d_stacks);
  thrust::device_ptr<uint3float1> dt_stacks_compacted = thrust::device_pointer_cast(d_stacks_compacted);

  thrust::copy_if(dt_stacks, dt_stacks + params.maxN *tsize.x*tsize.y*tsize.z, dt_stacks_compacted, is_not_empty());
  d_stacks_compacted = thrust::raw_pointer_cast(dt_stacks_compacted);
  std::cout << "+++++++++++++++++++++++" << std::endl;

  uint3float1* tmp = d_stacks;
  d_stacks = d_stacks_compacted;
  checkCudaErrors(hipFree(tmp));
  k_debug_lookup_stacks << <1, 1 >> >(d_stacks, tsize.x*tsize.y*tsize.z);
  hipDeviceSynchronize();

  // Allocate memory for gathered stacks uchar
  checkCudaErrors(hipMalloc((void**)&d_gathered4dstack, sizeof(float)*(sum*params.patch_size*params.patch_size*params.patch_size)));
  std::cout << "Allocated " << sizeof(float)*(sum*params.patch_size*params.patch_size*params.patch_size) << " bytes for gathered4dstack" << std::endl;

  k_gather_cubes << < 20, 256 >> > (img, size, params, d_stacks, d_nstacks, sum, d_gathered4dstack, d_nstacks_pow);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  
}

__global__ void dct3d(float* d_gathered4dstack, int patch_size){
  int x = threadIdx.x;
  int y = threadIdx.y;
  int z = threadIdx.z;
  int cuIdx = blockIdx.x;
  int stride = patch_size*patch_size*patch_size;
  // DCT 4x4 matrix
  const float dct_coeff[4][4] =
  {
    { 0.500000000000000f,  0.500000000000000f,  0.500000000000000f,  0.500000000000000f },
    { 0.653281482438188f,  0.270598050073099f, -0.270598050073099f, -0.653281482438188f },
    { 0.500000000000000f, -0.500000000000000f, -0.500000000000000f,  0.500000000000000f },
    { 0.270598050073099f, -0.653281482438188f,  0.653281482438188f, -0.270598050073099f }
  };
  const float dct_coeff_T[4][4] =
  {
    { 0.500000000000000f, 0.653281482438188f, 0.500000000000000f, 0.270598050073099f },
    { 0.500000000000000f, 0.270598050073099f, -0.500000000000000f, -0.653281482438188f },
    { 0.500000000000000f, -0.270598050073099f, -0.500000000000000f, 0.653281482438188f },
    { 0.500000000000000f, -0.653281482438188f, 0.500000000000000f, -0.270598050073099f }
  };
  // Load corresponding cube to the shared memory
  __shared__ float cube[4][4][4];
  int idx = (cuIdx*stride)+(x + y*patch_size + z*patch_size*patch_size);
  cube[z][y][x] = d_gathered4dstack[idx];
  __syncthreads();
  // Do 2d dct for rows (by taking slices along z direction)
  float tmp = dct_coeff[y][0] * cube[z][0][x] + dct_coeff[y][1] * cube[z][1][x] + dct_coeff[y][2] * cube[z][2][x] + dct_coeff[y][3] * cube[z][3][x];
  __syncthreads();
  cube[z][y][x] = tmp;
  __syncthreads();
  tmp = dct_coeff_T[0][x] * cube[z][y][0] + dct_coeff_T[1][x] * cube[z][y][1] + dct_coeff_T[2][x] * cube[z][y][2] + dct_coeff_T[3][x] * cube[z][y][3];
  __syncthreads();
  cube[z][y][x] = tmp;
  __syncthreads();
  // Grab Z vector
  float z_vec[4];
  for (int i = 0; i < 4; ++i){
    z_vec[i] = cube[i][y][x];
  }
  __syncthreads();
  cube[z][y][x] = z_vec[0] * dct_coeff[z][0] + z_vec[1] * dct_coeff[z][1] + z_vec[2] * dct_coeff[z][2] + z_vec[3] * dct_coeff[z][3];
  __syncthreads();
  d_gathered4dstack[idx] = cube[z][y][x];
}

void run_dct3d(float* d_gathered4dstack, uint gathered_size, int patch_size){
  dct3d << <gathered_size, dim3(4, 4, 4) >> > (d_gathered4dstack, patch_size);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}

__global__ void idct3d(float* d_gathered4dstack, int patch_size){
  int x = threadIdx.x;
  int y = threadIdx.y;
  int z = threadIdx.z;
  int cuIdx = blockIdx.x;
  int stride = patch_size*patch_size*patch_size;
  // DCT 4x4 matrix
  const float dct_coeff[4][4] =
  {
    { 0.500000000000000f, 0.500000000000000f, 0.500000000000000f, 0.500000000000000f },
    { 0.653281482438188f, 0.270598050073099f, -0.270598050073099f, -0.653281482438188f },
    { 0.500000000000000f, -0.500000000000000f, -0.500000000000000f, 0.500000000000000f },
    { 0.270598050073099f, -0.653281482438188f, 0.653281482438188f, -0.270598050073099f }
  };
  const float dct_coeff_T[4][4] =
  {
    { 0.500000000000000f, 0.653281482438188f, 0.500000000000000f, 0.270598050073099f },
    { 0.500000000000000f, 0.270598050073099f, -0.500000000000000f, -0.653281482438188f },
    { 0.500000000000000f, -0.270598050073099f, -0.500000000000000f, 0.653281482438188f },
    { 0.500000000000000f, -0.653281482438188f, 0.500000000000000f, -0.270598050073099f }
  };
  // Load corresponding cube to the shared memory
  __shared__ float cube[4][4][4];
  int idx = (cuIdx*stride) + (x + y*patch_size + z*patch_size*patch_size);
  cube[z][y][x] = d_gathered4dstack[idx];
  __syncthreads();
  float z_vec[4];
  for (int i = 0; i < 4; ++i){
    z_vec[i] = cube[i][y][x];
  }
  __syncthreads();
  cube[z][y][x] = z_vec[0] * dct_coeff_T[z][0] + z_vec[1] * dct_coeff_T[z][1] + z_vec[2] * dct_coeff_T[z][2] + z_vec[3] * dct_coeff_T[z][3];
  __syncthreads();
  float tmp = dct_coeff_T[y][0] * cube[z][0][x] + dct_coeff_T[y][1] * cube[z][1][x] + dct_coeff_T[y][2] * cube[z][2][x] + dct_coeff_T[y][3] * cube[z][3][x];
  __syncthreads();
  cube[z][y][x] = tmp;
  tmp = dct_coeff[0][x] * cube[z][y][0] + dct_coeff[1][x] * cube[z][y][1] + dct_coeff[2][x] * cube[z][y][2] + dct_coeff[3][x] * cube[z][y][3];
  __syncthreads();
  cube[z][y][x] = tmp;
  __syncthreads();
  d_gathered4dstack[idx] = cube[z][y][x];
}

void run_idct3d(float* d_gathered4dstack, uint gathered_size, int patch_size){
  idct3d << <gathered_size, dim3(4, 4, 4) >> > (d_gathered4dstack, patch_size);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
}

// (a,b) -> (a+b,a-b) without overflow
__device__ __host__ void whrotate(float& a, float& b)
{
  float t;
  t = a;
  a = a + b;
  b = t - b;
}

// Integer log2
__device__ __host__ long ilog2(long x)
{
  long l2 = 0;
  for (; x; x >>= 1) ++l2;
  return l2;
}

/**
* Fast Walsh-Hadamard transform
*/
__device__ __host__ void fwht(float* data, int size)
{
  const long l2 = ilog2(size) - 1;
  for (long i = 0; i < l2; ++i)
  {
    for (long j = 0; j < (1 << l2); j += 1 << (i + 1))
      for (long k = 0; k < (1 << i); ++k)
        whrotate(data[j + k], data[j + k + (1 << i)]);
  }
}

__global__ void k_run_wht_ht_iwht(float* d_gathered4dstack, 
                                  uint gathered_size, 
                                  int patch_size, 
                                  uint* d_nstacks_pow, 
                                  uint* accumulated_nstacks, 
                                  float* group_weights,
                                  int* group_keys){
  int x = threadIdx.x;
  int y = threadIdx.y;
  int z = threadIdx.z;
  int cuIdx = blockIdx.x;
  int stride = patch_size*patch_size*patch_size;
  float group_vector[16];
  int size = d_nstacks_pow[cuIdx];
  int group_start = accumulated_nstacks[cuIdx];
  //printf("\nSize: %d Group start: %d \n", size, group_start);

  for (int i = 0; i < size; i++){
    int gl_idx = (group_start*stride) + (x + y*patch_size + z*patch_size*patch_size + i*stride);
    group_vector[i] = d_gathered4dstack[gl_idx]; 
  }
  fwht(group_vector, size);
  //// Threshold
  float threshold = 2.7 * sqrtf((float)size);
  group_weights[cuIdx*stride + x + y*patch_size + z*patch_size*patch_size] = 0;
  group_keys[cuIdx*stride + x + y*patch_size + z*patch_size*patch_size] = cuIdx+1;
  for (int i = 0; i < size; i++){
    group_vector[i] /= size; // normalize
    if (fabs(group_vector[i]) > threshold)
    {
      group_weights[cuIdx*stride + x + y*patch_size + z*patch_size*patch_size] += 1;
    }
    else {
      group_vector[i] = 0;
    }
  }
  //// Inverse fwht
  fwht(group_vector, size);
  for (int i = 0; i < size; i++){
    int gl_idx = (group_start*stride) + (x + y*patch_size + z*patch_size*patch_size + i*stride);
    d_gathered4dstack[gl_idx] = group_vector[i];
  }
}

void run_wht_ht_iwht(float* d_gathered4dstack, uint gathered_size, int patch_size, uint* d_nstacks_pow, const uint3 tsize){
  uint* accumulated_nstacks;
  checkCudaErrors(hipMalloc((void **)&accumulated_nstacks, sizeof(uint)*gathered_size));
  thrust::device_ptr<uint> dt_accumulated_nstacks = thrust::device_pointer_cast(accumulated_nstacks);
  thrust::device_ptr<uint> dt_nstacks = thrust::device_pointer_cast(d_nstacks_pow);
  thrust::exclusive_scan(dt_nstacks, dt_nstacks + gathered_size, dt_accumulated_nstacks);
  accumulated_nstacks = thrust::raw_pointer_cast(dt_accumulated_nstacks);
  int groups = tsize.x*tsize.y*tsize.z;
  

  float* group_weights;
  int *group_keys, *dummy;
  checkCudaErrors(hipMalloc((void **)&group_weights, sizeof(float)*groups*patch_size*patch_size*patch_size)); // Cubes with weights for each group
  checkCudaErrors(hipMalloc((void **)&group_keys, sizeof(int)*groups*patch_size*patch_size*patch_size));
  checkCudaErrors(hipMalloc((void **)&dummy, sizeof(int)*groups*patch_size*patch_size*patch_size));

  k_run_wht_ht_iwht << <groups, dim3(4, 4, 4) >> > (d_gathered4dstack, gathered_size, patch_size, d_nstacks_pow, accumulated_nstacks, group_weights, group_keys);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());


  debug_kernel(group_weights);
  debug_kernel_int(group_keys);
  float* out_weights;
  checkCudaErrors(hipMalloc((void **)&out_weights, sizeof(float)*groups*patch_size*patch_size*patch_size));

  // Keys
  thrust::device_ptr<int> dt_dummy = thrust::device_pointer_cast(dummy);
  thrust::device_ptr<int> dt_group_keys = thrust::device_pointer_cast(group_keys);
  // Data
  thrust::device_ptr<float> dt_out_weights = thrust::device_pointer_cast(out_weights);
  thrust::device_ptr<float> dt_group_weights = thrust::device_pointer_cast(group_weights);

  //thrust::reduce_by_key(dt_group_keys, dt_group_keys + 64, dt_group_weights, dt_dummy, dt_out_weights);
  out_weights = thrust::raw_pointer_cast(dt_out_weights);
  debug_kernel(out_weights);
  checkCudaErrors(hipFree(accumulated_nstacks));
  checkCudaErrors(hipFree(group_weights));
  checkCudaErrors(hipFree(dummy));
  checkCudaErrors(hipFree(group_keys));

}
__global__ void k_aggregation(float* d_denoised_volume, 
                              float* d_weights_volume,
                            const uint3 size,
                            const uint3 tsize,
                            float* d_gathered4dstack, 
                            uint3float1* d_stacks, 
                            uint* d_nstacks, 
                            float* group_weights, 
                            const Parameters params){

  uint array_size = (tsize.x*tsize.y*tsize.z);
  for (int i = blockIdx.x*blockDim.x + threadIdx.x; i < array_size; i += blockDim.x*gridDim.x){

    uint3float1 ref = d_stacks[i];
    float weight = group_weights[i];
    int cube_size = params.patch_size*params.patch_size*params.patch_size;

    for (int z = 0; z < params.patch_size; ++z)
      for (int y = 0; y < params.patch_size; ++y)
        for (int x = 0; x < params.patch_size; ++x){
          int rx = x + ref.x;
          int ry = y + ref.y;
          int rz = z + ref.z;

          if (rx < 0 || rx >= size.x) continue;
          if (ry < 0 || ry >= size.y) continue;
          if (rz < 0 || rz >= size.z) continue;

          int img_idx = (rx)+(ry)*size.x + (rz)*size.x*size.y;
          int stack_idx = i*cube_size + (x)+(y + z*params.patch_size)*params.patch_size;
          //d_denoised_volume[img_idx] = d_gathered4dstack[stack_idx];
          atomicAdd(d_denoised_volume + img_idx, d_gathered4dstack[stack_idx]);
          atomicAdd(d_weights_volume + img_idx, weight);
        }
  }
}

__global__ void k_normalizer(float* d_denoised_volume,
                             const float* __restrict d_weights_volume,
                             const uint3 size)
{
  for (int Idz = blockDim.z * blockIdx.z + threadIdx.z; Idz < size.z; Idz += blockDim.z*gridDim.z)
    for (int Idy = blockDim.y * blockIdx.y + threadIdx.y; Idy < size.y; Idy += blockDim.y*gridDim.y)
      for (int Idx = blockDim.x * blockIdx.x + threadIdx.x; Idx < size.x; Idx += blockDim.x*gridDim.x)
      {
        int idx = Idx + Idy*size.x + Idx*size.x*size.y;
        float tmp = d_denoised_volume[idx];
        __syncthreads();
        d_denoised_volume[idx] = d_denoised_volume[idx] / d_weights_volume[idx];
      }
}

void run_aggregation(float* final_image,
                     const uint3 size, 
                     const uint3 tsize, 
                     float* d_gathered4dstack, 
                     uint3float1* d_stacks, 
                     uint* d_nstacks, 
                     float* group_weights,
                     const Parameters params)
{
  int im_size = size.x*size.y*size.z;
  int groups = tsize.x*tsize.y*tsize.z;
  float* d_junk_weights;
  float* junk_weights = new float[groups];
  for (int i = 0; i < groups; ++i) { junk_weights[i] = 1.0; }
  checkCudaErrors(hipMalloc((void **)&d_junk_weights, sizeof(float)*groups));
  checkCudaErrors(hipMemcpy(d_junk_weights, junk_weights, sizeof(float)*groups, hipMemcpyHostToDevice));

  float* d_denoised_volume, *d_weights_volume;
  checkCudaErrors(hipMalloc((void **)&d_denoised_volume, sizeof(float)*size.x*size.y*size.z));
  checkCudaErrors(hipMalloc((void **)&d_weights_volume, sizeof(float)*size.x*size.y*size.z));

  k_aggregation << <20, 1024 >> >(d_denoised_volume, d_weights_volume, size, tsize, d_gathered4dstack, d_stacks, d_nstacks, d_junk_weights, params);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
  k_normalizer << <20, dim3(32, 32, 1) >> >(d_denoised_volume, d_weights_volume, size);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());

  checkCudaErrors(hipMemcpy(final_image, d_denoised_volume, sizeof(float)*im_size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(d_junk_weights));
  delete[] junk_weights;
}